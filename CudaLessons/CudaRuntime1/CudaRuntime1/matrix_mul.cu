﻿#include "hip/hip_runtime.h"

#include "time.h"
#include "stdio.h"
#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

__global__ void matrixMul(int *a, int *b, int *c, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int row_offset = row * N;

	if (row < N && col < N) {
		int tmp = 0;

		for (int i = 0; i < N; i++) {
			tmp += a[row * N + i] * b[col + i * N];
		}

		c[row_offset + col] = tmp;


	}
	 


	// unique thread id
	//printf("\n block idx x %d", blockIdx.x);
	//printf("\n block idx y %d", blockIdx.y);

	//printf("\n thread idx x %d", threadIdx.x);
	//printf("\n thread idx y %d", threadIdx.y);


}

void verify_res(int* a, int* b, int* c, int N) {
	int tmp;
	for (int i = 0; i < N; i++) 
		printf("\n a %d", a[i]);

	for (int i = 0; i < N; i++)
		printf("\n b %d", b[i]);

    // row 
	for (int i = 0; i < N; i++) {
		// col 
		for (int j = 0; j < N; j++) {



			tmp = 0;
			// every el in row col pair
			for (int k = 0; k < N; k++) {

				tmp += a[i * N + k] * b[k  * N + j];

			}

			//assert(tmp == c[i * N + j]);
		}
	}
}

void init_matrix(int *m, int N) {
	for (int i = 0; i < N * N; i++) {
		m[i] = rand() % 100;
	}
}

int main() {

	int N = 2; //1 << 10;

	size_t bytes = N * N * sizeof(int);

	int *a, *b, *c;

	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);

	// Initialize our matricies

	init_matrix(a, N);
	init_matrix(b, N);

	//int threads = 16; 
	//int blocks = (N + threads - 1) / threads;

	//dim3 THREADS(threads, threads);
	//dim3 BLOCKS(blocks, blocks);

	//
	//matrixMul <<<BLOCKS, THREADS>>> (a, b, c, N);
	//cudaDeviceSynchronize();

	verify_res(a, b, c, N);


	cout << "Copmpleted" << endl;
	//int N = 1 << 9; 

	//double *x, *y, *ans;

	//cudaMalloc(&x, sizeof(double) * N * N);
	//cudaMalloc(&y, sizeof(double) * N * N);
	//cudaMalloc(&ans, sizeof(double) * N * N);

	//for (int i = 0; i < N; i++) {
	//	for (int j = 0; j < N; j++) {
	//		
	//		x[i * N + j] = 5;
	//		y[i * N + j] = 3;
	//	}
	//}


	//cudaFree(x);
	//cudaFree(y);
	//cudaFree(ans);

	return 0;
}