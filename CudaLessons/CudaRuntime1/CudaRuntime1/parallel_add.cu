﻿#include "stdio.h"
#include <cstdlib>
#include "time.h";

#include "hip/hip_runtime.h"



using namespace std;

__global__ 
void add2(int N, float *X, float *Y) {
	int t = threadIdx.x;
	int T = blockDim.x;

	for (int i = t; i < N; i += T) {
		Y[i] = X[i] + Y[i];
	}

	//printf("thread dim %d", threadIdx.x);
}


//int main() {
//
//	int N = 1 << 20;
//
//	printf("N %i", N);
//	
//	float *X, *Y;
//
//	cudaMallocManaged(&X, sizeof(float) * N);
//	cudaMallocManaged(&Y, sizeof(float) * N);
//
//	for (int i = 0; i < N; i++) {
//		X[i] = 1.0f;
//		Y[i] = 2.0f;
//	}
//
//	clock_t t;
//
//	printf("Start");
//
//	for (int i = 0; i < 512; i += 32) {
//		
//		int T = max(i, 1);
//
//		double avg = 0;
//
//		//for (int j = 0; j <= 10; j++) {
//		t = clock();
//
//		add2 << <dim3(1, 1, 1), dim3(T, 1, 1) >> > (N, X, Y);
//
//		cudaDeviceSynchronize();
//
//		t = clock() - t;
//		float r_t = float(t) * CLOCKS_PER_SEC / 1000;
//		printf("\n r_t %f th %d", r_t, i);
//		//}
//	}
//
//	cudaFree(X);
//	cudaFree(Y);
//
//	return 0;
//}