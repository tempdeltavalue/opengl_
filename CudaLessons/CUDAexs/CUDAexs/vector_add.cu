
#include <hip/hip_runtime.h>
﻿//#include "stdio.h"
//#include "cuda.h"
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//
//#define N (33 * 1024)
//
//__global__ void add(int* a, int* b, int* c) {
//	int tid = threadIdx.x + blockIdx.x * blockDim.x;
//
//	while (tid < N) {
//		c[tid] = a[tid] + b[tid];
//		tid += blockDim.x * gridDim.x;
//	}
//}
//
////int main() {
////	int a[N], b[N], c[N];
////	int* dev_a, * dev_b, * dev_c;
////
////	int arr_size = N * sizeof(int);
////
////	cudaMalloc(&dev_a, arr_size);
////	cudaMalloc(&dev_b, arr_size);
////	cudaMalloc(&dev_c, arr_size);
////
////	for (int i = 0; i < N; i++) {
////		a[i] = i;
////		b[i] = i * i;
////	}
////
////	cudaMemcpy(dev_a, a, arr_size, cudaMemcpyHostToDevice);
////	cudaMemcpy(dev_b, b, arr_size, cudaMemcpyHostToDevice);
////	cudaMemcpy(dev_c, c, arr_size, cudaMemcpyHostToDevice);
////
////	add << <128, 128 >> > (dev_a, dev_b, dev_c);
////
////	cudaMemcpy(c, dev_c, arr_size, cudaMemcpyDeviceToHost);
////
////	for (int i = 0; i < N; i++) {
////		printf("c %d", c[i]);
////	}
////
////	cudaFree(dev_a);
////	cudaFree(dev_b);
////	cudaFree(dev_c);
////
////	return 0;
////}