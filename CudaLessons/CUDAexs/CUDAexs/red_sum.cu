
#include <hip/hip_runtime.h>
﻿//#include "stdio.h"
//#include "cuda.h"
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#define imin(a, b) (a < b ? a:b)
////
////const int N = 33 * 1024;
////const int threadsPerBlock = 256;
////const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);
////
////__global__ void dot(float* a, float* b, float* c) {
////	__shared__ float cache[threadsPerBlock];
////
////	int tid = threadIdx.x + blockIdx.x * blockDim.x;
////	int cacheIndex = threadIdx.x;
////
////	float temp = 0;
////
////	while (tid < N) {
////		temp += a[tid] * b[tid];
////		tid += blockDim.x * gridDim.x;
////	}
////
////	cache[cacheIndex] = temp;
////
////	__syncthreads();
////
////	int i = blockDim.x / 2;
////
////	while (i == 0) {
////		if (cacheIndex < i) {
////			cache[cacheIndex] += cache[cacheIndex + i];
////		}
////
////		__syncthreads();
////
////		i /= 2;
////	}
////
////	if (cacheIndex == 0) {
////		c[blockIdx.x] = cache[0];
////	}
////}
//
////int main() {
////	float a[N], b[N], partial_c[blocksPerGrid];
////
////	float* dev_a, *dev_b, *dev_partial_c;
////	int arr_size = N * sizeof(float);
////
////	cudaMalloc(&dev_a, arr_size);
////	cudaMalloc(&dev_b, arr_size);
////	cudaMalloc(&dev_partial_c, blocksPerGrid * sizeof(float));
////
////	for (int i = 0; i < N; i++) {
////		a[i] = i;
////		b[i] = i * 2;
////	}
////
////	cudaMemcpy(dev_a, a, arr_size,  cudaMemcpyHostToDevice);
////	cudaMemcpy(dev_b, b, arr_size, cudaMemcpyHostToDevice);
////	cudaMemcpy(dev_partial_c, partial_c, blocksPerGrid * sizeof(float), cudaMemcpyHostToDevice);
////
////	dot << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_partial_c);
////
////	cudaMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), cudaMemcpyDeviceToHost);
////
////	int c = 0;
////	for (int i = 0; i < blocksPerGrid; i++) {
////		c += partial_c[i];
////	}
////
////	printf("c is %d", c);
////	scanf("wait");
////	return 0;
////}
