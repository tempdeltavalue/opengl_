
#include <hip/hip_runtime.h>
﻿//#include "stdio.h"
//#include "cuda.h"
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include "math.h"
//#include "common/cpu_bitmap.h"
//
//#define INF 2e10f
//#define rnd(x) (x * rand() / RAND_MAX)
//#define SPHERES 100
//#define DIM 1024
//
//
//
//struct Sphere
//{
//	float r, g, b;
//	float radius;
//	float x, y, z;
//
//	__device__ float hit(float ox, float oy, float* n) {
//		float dx = ox - x;
//		float dy = oy - y;
//
//		//if (dx * dx + dy * dy < radius * radius) {
//			float dz = sqrtf(radius * radius - dx * dx - dy * dy);
//			*n = dz / sqrtf(radius * radius);
//
//			return dz + z;
//		//}
//
//		//return -INF;
//	}
//};
//
//
//__global__ void kernel(Sphere* s, unsigned char* ptr) {
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	float ox = (x - DIM / 2);
//	float oy = (y - DIM / 2);
//
//	float r = 0, g = 0, b = 0;
//	float maxz = -INF;
//
//	for (int i = 0; i < SPHERES; i++) {
//		float n;
//		float t = s[i].hit(ox, oy, &n);
//		if (t > maxz) {
//			r = s[i].r * n;
//			g = s[i].g * n;
//			b = s[i].b * n;
//		}
//
//		ptr[offset * 4 + 0] = (int)(r * 255);
//		ptr[offset * 4 + 1] = (int)(g * 255);
//		ptr[offset * 4 + 2] = (int)(b * 255);
//		ptr[offset * 4 + 3] = 255;
//
//	}
//}
//
//int main() {
//	Sphere* s;
//
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//	cudaEventRecord(start, 0);
//
//	CPUBitmap bitmap(DIM, DIM);
//	unsigned char* dev_bitmap;
//
//	cudaMalloc(&dev_bitmap, bitmap.image_size());
//	cudaMalloc(&s, sizeof(Sphere) * SPHERES);
//
//	Sphere* temp_s = new Sphere[SPHERES];
//
//	for (int i = 0; i < SPHERES; i++) {
//		temp_s[i].r = rnd(1.0f);
//		temp_s[i].g = rnd(1.0f);
//		temp_s[i].b = rnd(1.0f);
//		temp_s[i].x = rnd(1000.0f);
//		temp_s[i].y = rnd(1000.0f);
//		temp_s[i].z = rnd(1000.0f);
//		temp_s[i].radius = rnd(100.0f) + 20;
//	}
//
//	 cudaMemcpy(s, temp_s, sizeof(Sphere) * SPHERES, cudaMemcpyHostToDevice);
//	//cudaMemcpyToSymbol(s, temp_s, sizeof(Sphere) * SPHERES);
//	
//	free(temp_s);
//
//	dim3 grid(DIM / 16, DIM / 16);
//	dim3 threads(16, 16);
//	kernel << <grid, threads >> > (s, dev_bitmap);
//	cudaMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), cudaMemcpyDeviceToHost);
//
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//
//	float elapsedTime;
//	cudaEventElapsedTime(&elapsedTime, start, stop);
//	printf("Time to generate %3.1f ms", elapsedTime);
//
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	bitmap.display_and_exit();
//
//	cudaFree(dev_bitmap);
//	cudaFree(s);
//}
//
