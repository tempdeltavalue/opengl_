﻿/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */


  #include "../common/book.h"
  #include "../common/cpu_bitmap.h"

#define dim 1000

struct cucomplex {
    float   r;
    float   i;
    cucomplex(float a, float b) : r(a), i(b) {}
    float magnitude2(void) { return r * r + i * i; }
    cucomplex operator*(const cucomplex& a) {
        return cucomplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    cucomplex operator+(const cucomplex& a) {
        return cucomplex(r + a.r, i + a.i);
    }
};

int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * (float)(dim / 2 - x) / (dim / 2);
    float jy = scale * (float)(dim / 2 - y) / (dim / 2);

    cucomplex c(-0.8, 0.156);
    cucomplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

void kernel(unsigned char* ptr) {
    for (int y = 0; y < dim; y++) {
        for (int x = 0; x < dim; x++) {
            int offset = x + y * dim;

            int juliavalue = julia(x, y);
            ptr[offset * 4 + 0] = 255 * juliavalue;
            ptr[offset * 4 + 1] = 0;
            ptr[offset * 4 + 2] = 0;
            ptr[offset * 4 + 3] = 255;
        }
    }
}

int main(void) {
    CPUBitmap bitmap(dim, dim);
    unsigned char* ptr = bitmap.get_ptr();

    kernel(ptr);

    bitmap.display_and_exit();
}